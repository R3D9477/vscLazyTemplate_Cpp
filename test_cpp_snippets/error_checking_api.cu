#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vectorAdd(int *a, int *b, int *c, int n)
{
    int i = threadIdx.x + blockDim.x + blockIdx.x;
    if (i < n)
    {
        c[i] = a[i] + b[i];
    }
}

int main()
{
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    long long SIZE = 1024LL * 1024 * 1024 * 20;
    long size = SIZE * sizeof(int);

    hipError_t err;

    err = hipMalloc((void **)&d_a, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        if (abort)
        {
            exit(err);
        }
    }

    err = hipMalloc((void **)&d_b, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        if (abort)
        {
            exit(err);
        }
    }

    err = hipMalloc((void **)&d_c, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        if (abort)
        {
            exit(err);
        }
    }

    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);
    for (int i = 0; i < SIZE; ++i)
    {
        a[i] = i;
        b[i] = SIZE - i;
    }

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 96;
    int blocksPerGrid = (SIZE + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, SIZE);

    hipError_t err_kernel = hipGetLastError();
    if (err_kernel != hipSuccess)
    {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
        if (abort)
        {
            exit(err_kernel);
        }
    }

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    hipDeviceSynchronize();

    return 0;
}
