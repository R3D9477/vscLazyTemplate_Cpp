#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <stdio.h>

#include "cppinfo.hpp"
#include "project_version.hpp"

__global__ void gridInfo()
{
    int warp_id = threadIdx.x / 32;
    printf("Hello from block ID %d, thread ID %d, warp ID %d\n", blockIdx.x, threadIdx.x, warp_id);
}

int main(void)
{
    {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);

        gridInfo<<<2, 64>>>();
        hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float ms = 0;
        hipEventElapsedTime(&ms, start, stop);
        std::cout << "CUDA kernel execution time: " << ms << "ms" << std::endl;
    }

    {
        std::cout << "Project version: " << project_version() << std::endl;

        CppInfo cppInfo;
        std::cout << "C++ language version: " << cppInfo.GetLanguageVersion() << std::endl;
        std::cout << "C++ compiler version: " << cppInfo.GetCompilerVersion() << std::endl;
    }

    return 0;
}
